#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include ""
#include "math_functions.hpp"

__global__ void partedCholesky_GPU(int M, float* A)
{
	int x = threadIdx.x+blockIdx.x*1024;
	float multi = 0;

	//Left part
	int j = 0;
	for (; j < 9 * M; j++)
	{
		A[(x + 9 * M) * 18 * M + j] = (A[(x + 9 * M) * 18 * M + j]-multi)/A[j*18*M+j];

		multi = 0;
		for (int i = 0; i <= j; i++)
		{
			multi += A[(j+1) * 18 * M + i] * A[(x + 9 * M) * 18 * M + i];
		}
	}

	__syncthreads();

	multi = 0;
	for (int i = 0; i <= j; i++)
	{
		multi += A[j * 18 * M + i] * A[(x + 9 * M) * 18 * M + i];
	}


	//Right part
	for (j = 9 * M; j < 18 * M; j++)
	{
		if (j - 9 * M == x)
		{
			A[(x + 9 * M) * 18 * M + j] = sqrt(A[(x + 9 * M) * 18 * M + j] - multi);
			break;
		}
		A[(x + 9 * M) * 18 * M + j] = (A[(x + 9 * M) * 18 * M + j] - multi) / A[j * 18 * M + j];
		
		multi = 0;
		for (int i = 0; i <= j; i++)
		{
			multi += A[(j+1) * 18 * M + i] * A[(x + 9 * M) * 18 * M + i];
		}

		//synchronize every column
		__syncthreads();
	}

	__syncthreads();


}


void partedCholesky(int M, float* A)
{
	int size = 18 * M;
	const dim3 blockSize(1024);
	const dim3 gridSize(18*M/1024+1);

	partedCholesky_GPU << <gridSize, blockSize >> >(M, A);

}